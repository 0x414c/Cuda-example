#include <assert.h>  // assert
#include <float.h>  // DECIMAL_DIG
#include <stdio.h>  // fprintf, printf, stderr
#include <stddef.h>  // size_t, NULL
#include <stdlib.h>  // free, malloc, EXIT_SUCCESS
#include <math.h>  // cos, fabs, isfinite, sin

#include "hip/hip_runtime.h"  // __global__, __restrict__, cuda*


#ifndef DECIMAL_DIG
#define DECIMAL_DIG (21)
#endif  // DECIMAL_DIG


typedef double real_type;


__global__ void
vectorAddition (
  const real_type * __restrict__ a, const real_type * __restrict__ b, real_type * __restrict__ c, size_t count
)
{
  const unsigned thread_id = blockDim.x * blockIdx.x + threadIdx.x;
  if (thread_id < count)
  {
    c [thread_id] = a [thread_id] + b [thread_id];
  }
}


int
isClose (real_type x, real_type y, real_type rel_tol, real_type abs_tol)
{
  assert (rel_tol >= 0.0);
  assert (abs_tol >= 0.0);

  if (isfinite (x) && isfinite (y))
  {
    if (x == y)
    {
      return 1;
    }

    const real_type abs_diff (fabs (x - y));

    return (abs_diff <= abs_tol)
        || (abs_diff <= (rel_tol * fabs (x)))
        || (abs_diff <= (rel_tol * fabs (y)));
  }

  return x == y;
}


int
main (int argc, char * argv [])
{
  const size_t count = 65536;
  const size_t bytes = count * sizeof (real_type);
  real_type * __restrict__ const host_a = (real_type *) malloc (bytes);
  if (host_a == NULL)
  {
    fprintf (stderr, "Error: couldn't allocate memory for host vector `a' (%zu bytes).\n", bytes);

    return EXIT_FAILURE;
  }

  real_type * __restrict__ const host_b = (real_type *) malloc (bytes);
  if (host_b == NULL)
  {
    fprintf (stderr, "Error: couldn't allocate memory for host vector `b' (%zu bytes).\n", bytes);

    return EXIT_FAILURE;
  }

  real_type * __restrict__ const host_c = (real_type *) malloc (bytes);
  if (host_c == NULL)
  {
    fprintf (stderr, "Error: couldn't allocate memory for host vector `c' (%zu bytes).\n", bytes);

    return EXIT_FAILURE;
  }

  real_type * __restrict__ const device_a = NULL;
  hipError_t allocated = hipMalloc ((void **) & device_a, bytes);
  if (allocated != hipSuccess)
  {
    fprintf (stderr, "Error: couldn't allocate memory for device vector `a' (%zu bytes): %s.\n", bytes, hipGetErrorString (allocated));

    return EXIT_FAILURE;
  }

  real_type * __restrict__ const device_b = NULL;
  allocated = hipMalloc ((void **) & device_b, bytes);
  if (allocated != hipSuccess)
  {
    fprintf (stderr, "Error: couldn't allocate memory for device vector `b' (%zu bytes): %s.\n", bytes, hipGetErrorString (allocated));

    return EXIT_FAILURE;
  }

  real_type * __restrict__ const device_c = NULL;
  allocated = hipMalloc ((void **) & device_c, bytes);
  if (allocated != hipSuccess)
  {
    fprintf (stderr, "Error: couldn't allocate memory for device vector `c' (%zu bytes): %s.\n", bytes, hipGetErrorString (allocated));

    return EXIT_FAILURE;
  }

  for (size_t i = 0; i < count; ++ i)
  {
/*    host_a [i] = i;
    host_b [i] = count - i - 1;*/
    host_a [i] = sin (i) * sin (i);
    host_b [i] = cos (i) * cos (i);
  }

  hipError_t copied = hipMemcpy (device_a, host_a, bytes, hipMemcpyHostToDevice);
  if (copied != hipSuccess)
  {
    fprintf (stderr, "Error: couldn't copy host vector `a' to device: %s.\n", hipGetErrorString (copied));

    return EXIT_FAILURE;
  }

  copied = hipMemcpy (device_b, host_b, bytes, hipMemcpyHostToDevice);
  if (copied != hipSuccess)
  {
    fprintf (stderr, "Error: couldn't copy host vector `b' to device: %s.\n", hipGetErrorString (copied));

    return EXIT_FAILURE;
  }

  const unsigned threads_per_block = 256;
  const unsigned blocks_per_grid = (count + threads_per_block - 1) / threads_per_block;
  hipGetLastError ();
  vectorAddition <<<blocks_per_grid, threads_per_block>>> (device_a, device_b, device_c, count);
  hipDeviceSynchronize ();
  const hipError_t added = hipGetLastError ();
  if (added != hipSuccess)
  {
    fprintf (stderr, "Error: couldn't launch kernel: %s.\n", hipGetErrorString (added));

    return EXIT_FAILURE;
  }

  copied = hipMemcpy (host_c, device_c, bytes, hipMemcpyDeviceToHost);
  if (copied != hipSuccess)
  {
    fprintf (stderr, "Error: couldn't copy vector `c' to host: %s.\n", hipGetErrorString (copied));

    return EXIT_FAILURE;
  }

  for (size_t i = 0; i < count; ++ i)
  {
    const real_type expected = host_a [i] + host_b [i];
    const real_type actual = host_c [i];
    if (! isClose (expected, actual, 1e-8, 1e-16))
    {
      fprintf (
        stderr, "Test failed at element %zu: expected=%.*f;actual=%.*f;.\n",
        i, DECIMAL_DIG, expected, DECIMAL_DIG, actual
      );

      return EXIT_FAILURE;
    }
  }

  hipError_t freed = hipFree (device_a);
  if (freed != hipSuccess)
  {
    fprintf (stderr, "Error: couldn't free device vector `a': %s.\n", hipGetErrorString (freed));

    return EXIT_FAILURE;
  }

  freed = hipFree (device_b);
  if (freed != hipSuccess)
  {
    fprintf (stderr, "Error: couldn't free device vector `b': %s.\n", hipGetErrorString (freed));

    return EXIT_FAILURE;
  }

  freed = hipFree (device_c);
  if (freed != hipSuccess)
  {
    fprintf (stderr, "Error: couldn't free device vector `c': %s.\n", hipGetErrorString (freed));

    return EXIT_FAILURE;
  }

  free (host_a);
  free (host_b);
  free (host_c);

  printf ("Done.\n");

  return EXIT_SUCCESS;
}
